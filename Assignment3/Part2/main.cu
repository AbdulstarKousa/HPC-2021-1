#include "hip/hip_runtime.h"
/* main.c - Poisson problem in 3D
 *
 */
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <hip/hip_runtime_api.h>
#include "alloc3d.h"
#include "alloc3d_gpu.h"
#include "transfer3d_gpu.h"
#include "print.h"
#include "sin_test.h"
#include "init.h"
#include "jacobi_gpu.h"

#ifdef _JACOBI
#include "jacobi.h"
#endif

#define N_DEFAULT 100

void warmUp()
{
    const int device = 0;
        
    // Wake up GPU from power save state.
    fflush(stdout);
    //double time = omp_get_wtime();
    hipSetDevice(device);           // Set the device to 0 or 1.
    double *dummy_d;
    hipMalloc((void**)&dummy_d, 0); // We force the creation of context on the
                                         // device by allocating a dummy variable.
    //printf("time = %lf seconds\n", omp_get_wtime() - time);
}

int
main(int argc, char *argv[]) {

    int 	N = N_DEFAULT;
    int 	iter_max = 1000;
    double	tolerance;
    double	start_T;
    int		output_type = 0;
    int     jacobi_type = 0; 
    char	*output_prefix = "poisson_res";
    char    *output_ext    = "";
    char	output_filename[FILENAME_MAX];
    double 	***h_u = NULL;
    double 	***h_f = NULL;
    double 	***h_u_next = NULL;
    double 	***d_u = NULL;
    double 	***d_f = NULL;
    double 	***d_u_next = NULL;
    int m;

    /* get the paramters from the command line */
    N         = atoi(argv[1]);	// grid size
    iter_max  = atoi(argv[2]);  // max. no. of iterations
    tolerance = atof(argv[3]);  // tolerance
    start_T   = atof(argv[4]);  // start T for all inner grid points
    if (argc == 6) {
    jacobi_type = atoi(argv[5]);  // exercise number 
    }
    if (argc == 7) {
    output_type = atoi(argv[6]);  // ouput type
    }

    printf("output type %d \n", output_type);


    //Allocate memory on HOST
    long long N2 = N + 2; 
    //printf("Allocating mem_space on CPU\n");

    if ( (h_u = d_malloc_3d(N2, N2, N2)) == NULL ) {
        perror("array h_u: allocation failed");
        exit(-1);
    }
    if ( (h_u_next = d_malloc_3d(N2, N2, N2)) == NULL ) {
        perror("array h_u_next: allocation failed");
        exit(-1);
    }
    if ( (h_f = d_malloc_3d(N2, N2, N2)) == NULL ) {
        perror("array h_f: allocation failed");
        exit(-1);
    }

    //Allocate memory on DEVICE
    if(jacobi_type == 31){ //skip if we need to GPU's 
        //printf("Skip allocating memory\n");
    }
    else{
        //printf("Allocating mem_space on GPU\n");

        if ( (d_u = d_malloc_3d_gpu(N2, N2, N2)) == NULL ) {
            perror("array d_u: allocation failed");
            exit(-1);
        }
        if ( (d_u_next = d_malloc_3d_gpu(N2, N2, N2)) == NULL ) {
            perror("array d_u_next: allocation failed");
            exit(-1);
        }
        if ( (d_f = d_malloc_3d_gpu(N2, N2, N2)) == NULL ) {
            perror("array d_f: allocation failed");
            exit(-1);
        }
    }


    // Call different exercises 
    switch(jacobi_type) {
        case 11:
            {
            //printf("\n");
            //printf("Jacopi GPU ex5\n");
 

            //Iniliazie matrices on HOST  
            //printf("Iniliazie matrices on HOST\n");
            init(h_f, h_u, h_u_next, N, start_T);  
  
            //warm up GPU
            warmUp();

            double time_t = omp_get_wtime();
            
            //Transfer data to DEVICE 
            //printf("Transfer data to DEVICE \n");
            transfer_3d(d_u, h_u, N2, N2, N2, hipMemcpyHostToDevice); 
            transfer_3d(d_u_next, h_u_next, N2, N2, N2, hipMemcpyHostToDevice); 
            transfer_3d(d_f, h_f, N2, N2, N2, hipMemcpyHostToDevice); 

            jacobi_gpu_wrap1(d_f,d_u,d_u_next,N,tolerance,iter_max,&m);
            //printf("Out of Jabobi\n");

            //printf("Transfer data back to HOST \n");
            transfer_3d(h_u,d_u, N2, N2, N2, hipMemcpyDeviceToHost); 
            
            printf("total time = %lf \n", (omp_get_wtime() - time_t));
            
            break;
            }
        
        case 12: //OBS HOW TO WE SAT MAKEFILE TO 1 CPU     numactl --cpunodebind=0 
            {
            //printf("\n");
            //printf("Jacopi CPU ex5\n");
            //Initialize matrices
            init(h_f, h_u, h_u_next, N, start_T);  

            //Call reference jacobi 
            //printf("Calling reference jacobi\n");
            double time_t1 = omp_get_wtime();

            jacobi_no_norm(h_f,h_u,h_u_next,N,tolerance,iter_max,&m);

            printf("total time = %lf \n", (omp_get_wtime() - time_t1));
            //printf("Out of reference jacobi\n");

            break;
            } 

        case 21:
            {
            //printf("\n");
            //printf("Jacopi GPU ex6\n");
            //warm up GPU
            warmUp(); 

            //Iniliazie matrices on HOST  
            //printf("Iniliazie matrices on HOST\n");
            init(h_f, h_u, h_u_next, N, start_T);  

            double time_t2 = omp_get_wtime();
            
            //Transfer data to DEVICE 
            //printf("Transfer data to DEVICE \n");
            transfer_3d(d_u, h_u, N2, N2, N2, hipMemcpyHostToDevice); 
            transfer_3d(d_u_next, h_u_next, N2, N2, N2, hipMemcpyHostToDevice); 
            transfer_3d(d_f, h_f, N2, N2, N2, hipMemcpyHostToDevice); 


            jacobi_gpu_wrap2(d_f,d_u,d_u_next,N,tolerance,iter_max,&m);
            //printf("Out of Jabobi exercise 6\n");

            //printf("Transfer data back to HOST \n");
            transfer_3d(h_u,d_u, N2, N2, N2, hipMemcpyDeviceToHost);  
            
            printf("total time = %lf\n", (omp_get_wtime() - time_t2));
            
            break;
            }
        case 31:
            {
            if( N % 2 != 0){
                printf("N IS NOT EVEN\n");
                perror("failed.......!");
                exit(-1);
            }

            //printf("\n");
            printf("Jacobi running two GPU ex7\n");

            //Allocate device memory 
            double 	***d0_u = NULL;
            double 	***d0_f = NULL;
            double 	***d0_u_next = NULL;

            double 	***d1_u = NULL;
            double 	***d1_f = NULL;
            double 	***d1_u_next = NULL;

            //Device 0 
            hipSetDevice(0);
            if ( (d0_u = d_malloc_3d_gpu(N2/2, N2, N2)) == NULL ) {
                perror("array d_u: allocation failed");
                exit(-1);
            }
            if ( (d0_u_next = d_malloc_3d_gpu(N2/2, N2, N2)) == NULL ) {
                perror("array d_u_next: allocation failed");
                exit(-1);
            }
            if ( (d0_f = d_malloc_3d_gpu(N2/2, N2, N2)) == NULL ) {
                perror("array d_f: allocation failed");
                exit(-1);
            }

            //Device 1
            hipSetDevice(1);
            if ( (d1_u = d_malloc_3d_gpu(N2/2, N2, N2)) == NULL ) {
                perror("array d_u: allocation failed");
                exit(-1);
            }
            if ( (d1_u_next = d_malloc_3d_gpu(N2/2, N2, N2)) == NULL ) {
                perror("array d_u_next: allocation failed");
                exit(-1);
            }
            if ( (d1_f = d_malloc_3d_gpu(N2/2, N2, N2)) == NULL ) {
                perror("array d_f: allocation failed");
                exit(-1);
            }

             //Allocate host memory 
             double 	***h0_u = NULL;
             double 	***h0_f = NULL;
             double 	***h0_u_next = NULL;      
             
             double 	***h1_u = NULL;
             double 	***h1_f = NULL;
             double 	***h1_u_next = NULL;      

             if ( (h0_u = d_malloc_3d(N2/2, N2, N2)) == NULL ) {
                 perror("array d_u: allocation failed");
                 exit(-1);
             }
             if ( (h0_u_next = d_malloc_3d(N2/2, N2, N2)) == NULL ) {
                 perror("array d_u_next: allocation failed");
                 exit(-1);
             }
             if ( (h0_f = d_malloc_3d(N2/2, N2, N2)) == NULL ) {
                 perror("array d_f: allocation failed");
                 exit(-1);
             }
             if ( (h1_u = d_malloc_3d(N2/2, N2, N2)) == NULL ) {
                 perror("array d_u: allocation failed");
                 exit(-1);
             }
             if ( (h1_u_next = d_malloc_3d(N2/2, N2, N2)) == NULL ) {
                 perror("array d_u_next: allocation failed");
                 exit(-1);
             }
             if ( (h1_f = d_malloc_3d(N2/2, N2, N2)) == NULL ) {
                 perror("array d_f: allocation failed");
                 exit(-1);
             }
            
             printf("Im here 0\n");

            //warm up GPU
            hipSetDevice(0);
            warmUp();
            hipSetDevice(1);
            warmUp();  

            //Iniliazie matrices on HOST  
            printf("Iniliazie matrices on HOST\n");
            init(h_f, h_u, h_u_next, N, start_T); 
            printf("Im here 1\n");

            for(int i = 0; i < N2; i++){
                for(int j = 0; j < N2; j++){
                    for(int k = 0; k < N2; k++){
                        if(i < N2/2){
                            h0_f[i][j][k] = h_f[i][j][k];
                            h0_u[i][j][k] = h_u[i][j][k];
                            h0_u_next[i][j][k] = h_u_next[i][j][k];  
                        }
                        else{
                            h1_f[i - (N2/2)][j][k] = h_f[i][j][k];
                            h1_u[i - (N2/2)][j][k] = h_u[i][j][k];
                            h1_u_next[i - (N2/2)][j][k] = h_u_next[i][j][k];
                        }
                    }
                }
            }
            
            double time_t2 = omp_get_wtime();
            
            //Transfer data to DEVICE 0 
            printf("Transfer data to DEVICE 0 \n");
            hipSetDevice(0);
            hipDeviceEnablePeerAccess(1, 0);
            transfer_3d(d0_u, h0_u, N2/2, N2, N2, hipMemcpyHostToDevice); 
            printf("Transfer data to DEVICE 0 \n");
            transfer_3d(d0_u_next, h0_u_next, N2/2, N2, N2, hipMemcpyHostToDevice); 
            printf("Transfer data to DEVICE 0 \n");
            transfer_3d(d0_f, h0_f, N2/2, N2, N2, hipMemcpyHostToDevice); 

            //Transfer data to DEVICE 1
            printf("Transfer data to DEVICE 1 \n");              
        
            hipSetDevice(1);
            hipDeviceEnablePeerAccess(0, 0);
            transfer_3d(d1_u, h1_u, N2/2, N2, N2, hipMemcpyHostToDevice); 
            transfer_3d(d1_u_next, h1_u_next, N2/2, N2, N2, hipMemcpyHostToDevice); 
            transfer_3d(d1_f, h1_f, N2/2, N2, N2, hipMemcpyHostToDevice);            


            jacobi_gpu_wrap3(d0_f,d0_u,d0_u_next,d1_f,d1_u,d1_u_next,N,tolerance,iter_max,&m);
            printf("Out of Jabobi exercise 7\n");

            //Transfer data back to HOST 
            printf("Transfer data back to HOST from DEVICE 0 \n");
            hipSetDevice(0);
            transfer_3d(h0_u, d0_u, N2/2, N2, N2, hipMemcpyDeviceToHost);  

            printf("Transfer data back to HOST from DEVICE 1 \n");
            hipSetDevice(1);
            transfer_3d(h1_u, d1_u, N2/2, N2, N2, hipMemcpyDeviceToHost);  

            
            printf("total time = %lf\n", (omp_get_wtime() - time_t2));


            for(int i = 0; i < N2; i++){
                for(int j = 0; j < N2; j++){
                    for(int k = 0; k < N2; k++){
                        if(i < N2/2){
                            h_f[i][j][k] = h0_f[i][j][k];
                            h_u[i][j][k] = h0_u[i][j][k];
                            h_u_next[i][j][k] = h0_u_next[i][j][k];  
                        }
                        else{
                            h_f[i][j][k] = h1_f[i- (N2/2)][j][k];
                            h_u[i][j][k] = h1_u[i- (N2/2)][j][k];
                            h_u_next[i][j][k] = h1_u_next[i- (N2/2)][j][k];
                        }
                    }
                }
            }

            free_gpu(d0_f); 
            free_gpu(d0_u); 
            free_gpu(d0_u_next); 
            free_gpu(d1_f); 
            free_gpu(d1_u); 
            free_gpu(d1_u_next); 

            free(h0_u);
            free(h0_u_next);
            free(h0_f);
            free(h1_u);
            free(h1_u_next);
            free(h1_f);

            
            break;
        }

        case 41:
        {
            printf("\n");
            printf("Jacopi running with norm GPU ex8\n");
            //warm up GPU
            warmUp(); 

            //Iniliazie matrices on HOST  
            printf("Iniliazie matrices on HOST\n");
            init(h_f, h_u, h_u_next, N, start_T);  

            double time_t2 = omp_get_wtime();
            
            //Transfer data to DEVICE 
            printf("Transfer data to DEVICE \n");
            transfer_3d(d_u, h_u, N2, N2, N2, hipMemcpyHostToDevice); 
            transfer_3d(d_u_next, h_u_next, N2, N2, N2, hipMemcpyHostToDevice); 
            transfer_3d(d_f, h_f, N2, N2, N2, hipMemcpyHostToDevice); 


            jacobi_gpu_wrap4new(d_f,d_u,d_u_next,N,tolerance,iter_max,&m);
            printf("Out of Jabobi exercise 8\n");

            printf("Transfer data back to HOST \n");
            transfer_3d(h_u,d_u, N2, N2, N2, hipMemcpyDeviceToHost);  
            
            printf("total time = %lf seconds, with N=%d and %d iterations to break out of %d iterations\n", (omp_get_wtime() - time_t2),N,m,iter_max);
            
            break;
        }
        case 42: //OBS HOW TO WE SAT MAKEFILE TO 1 CPU     numactl --cpunodebind=0 
        {
            printf("\n");
            printf("Jacopi CPU with norm ex8\n");
            //Initialize matrices
            init(h_f, h_u, h_u_next, N, start_T);  
            double norm_result = 0.0; 

            //Call reference jacobi 
            printf("Calling reference jacobi\n");
            double time_t1 = omp_get_wtime();

            norm_result = jacobi(h_f,h_u,h_u_next,N,tolerance,iter_max,&m);

            printf("total time = %lf seconds, with N=%d and %d iterations and breaks after %d \n", (omp_get_wtime() - time_t1),N,iter_max, m);
            printf("Out of reference jacobi\n");
 
        break;
        } 

        default:
        {
            fprintf(stderr, "No valid version specified!\n");
            break;
        }
    }





    /*
    int i,j,k; 
    for (i = 0; i < N+2; i++) {
        for (j = 0; j < N+2; j++) {
            for (k = 0; k < N+2; k++) { 
                printf("%lf ",h_u[i][j][k]);
            }
        }
        printf("\n");
    }
    */
   
    

/*
    //Iniliazie matrices 
    #ifdef _SIN_TEST
    printf("Running sin_test \n");
    sin_init(f, u, u_next, N);
    #else
    init(f, u, u_next, N, start_T);
    #endif
*/


    // dump  results if wanted 
    switch(output_type) {
	case 0:
	    // no output at all
	    break;
	case 3:
	    output_ext = ".bin";
	    sprintf(output_filename, "%s_%d%s", output_prefix, N+2, output_ext);
	    fprintf(stderr, "Write binary dump to %s: ", output_filename);
	    print_binary(output_filename, N+2, h_u);
	    break;
    case 4:
        printf("\n I'm here case 4");
	    output_ext = ".vtk";
	    sprintf(output_filename, "%s_%d%s", output_prefix, N+2, output_ext);
	    fprintf(stderr, "Write VTK file to %s: ", output_filename);
	    print_vtk(output_filename, N+2, h_u);
	    break;
	default:
	    fprintf(stderr, "Non-supported output type!\n");
	    break;
    }

    // de-allocate memory
    free(h_u);
    free(h_u_next);
    free(h_f);
    free_gpu(d_u);
    free_gpu(d_u_next);
    free_gpu(d_f);

    return(0);
}