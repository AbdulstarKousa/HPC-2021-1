#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>


__global__ 
void jacobi_kernel1(
    double*** d_f,        /* 3D matrix "Cube" of function values, Second derivatives of temperature  */
    double*** d_u,        /* 3D matrix "Cube" of temperature estimates */
    double *** d_u_next,  /* 3D matrix "Cube" to hold new temperature estimates */
    int N,                /* #nr. interior grid points */
    double d_squared, 
    double inv              ){

    int i,j,k; 
    int N2 = N + 2; 

    for (i = 1; i < N2 - 1; i++) {
        for (j = 1; j < N2 - 1; j++) {
            for (k = 1; k < N2 - 1; k++) {
                
                d_u_next[i][j][k] = inv * (d_u[i-1][j][k] + d_u[i+1][j][k] + d_u[i][j-1][k] + d_u[i][j+1][k] + d_u[i][j][k-1] + d_u[i][j][k+1] + d_squared * d_f[i][j][k]);
                    
            }
        }
    }
    //printf("Leaving kernel function\n");
}

void jacobi_gpu_wrap1(  double*** d_f,        /* 3D matrix "Cube" of function values, Second derivatives of temperature  */
                double*** d_u,        /* 3D matrix "Cube" of temperature estimates */
                double *** d_u_next,  /* 3D matrix "Cube" to hold new temperature estimates */
                int N,              /* #nr. interior grid points */
                double tolerance,   /* threshold */
                int iter_max,       /* maximum nr. of iterations */
                int * mp){           /* #nr. the iteration needed to get a suciently small diference*/

    double delta= (double)(2.0/((double)(N+1))); // the grid spacing.
    double d_squared = delta*delta;
    double inv = 1.0/6.0;
    double *** temp; // to swipe between u and u_next.
    int m = 0;

    //printf("Entering while loop\n");
    while (m < iter_max) 
    {

        jacobi_kernel1<<<1,1>>>(d_f, d_u, d_u_next, N, d_squared,inv);    
        hipDeviceSynchronize();          

        temp = d_u;
        d_u = d_u_next; 
        d_u_next = temp;
        
        m++;
    }

    //printf("End Jacobi wrapper\n");
}




/* *************
 EXERCISE 6 
************* */

__global__ 
void jacobi_kernel2(
    double*** d_f,        /* 3D matrix "Cube" of function values, Second derivatives of temperature  */
    double*** d_u,        /* 3D matrix "Cube" of temperature estimates */
    double *** d_u_next,  /* 3D matrix "Cube" to hold new temperature estimates */
    int N,                /* #nr. interior grid points */
    double d_squared, 
    double inv              ){

    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if(0 < i && 0 < j && 0 < k && i < N+1 && j < N+1 && k < N+1)
    {    
        d_u_next[i][j][k] = inv * (d_u[i-1][j][k] + d_u[i+1][j][k] + d_u[i][j-1][k] + d_u[i][j+1][k] + d_u[i][j][k-1] + d_u[i][j][k+1] + d_squared * d_f[i][j][k]);
    }
}

void jacobi_gpu_wrap2(  double*** d_f,        /* 3D matrix "Cube" of function values, Second derivatives of temperature  */
                double*** d_u,        /* 3D matrix "Cube" of temperature estimates */
                double *** d_u_next,  /* 3D matrix "Cube" to hold new temperature estimates */
                int N,              /* #nr. interior grid points */
                double tolerance,   /* threshold */
                int iter_max,       /* maximum nr. of iterations */
                int * mp){           /* #nr. the iteration needed to get a suciently small diference*/

    double delta= (double)(2.0/((double)(N+1))); // the grid spacing.
    double d_squared = delta*delta;
    double inv = 1.0/6.0;
    double *** temp; // to swipe between u and u_next.
    int m = 0;
    
    int threads_blck = 8; //optmized to be fastest with 8 threads per block (each dim)

    dim3 dimBlock(threads_blck,threads_blck,threads_blck);// threads per block
    dim3 dimGrid(((N+2)/dimBlock.x)+1,((N+2)/dimBlock.y)+1,((N+2)/dimBlock.z)+1); // xx blocks in total

    //printf("Entering while loop\n");
    while (m < iter_max) //&& norm_result > tolerance 
    {
        jacobi_kernel2<<<dimGrid,dimBlock>>>(d_f, d_u, d_u_next, N, d_squared,inv);    
        hipDeviceSynchronize();          

        temp = d_u;
        d_u = d_u_next; 
        d_u_next = temp;
        
        m++;
    }
}



/* *************
 EXERCISE 7 
************* */

__global__ 
void jacobi_kernel31(
    double*** d0_f,        /* 3D matrix "Cube" of function values, Second derivatives of temperature  */
    double*** d0_u,        /* 3D matrix "Cube" of temperature estimates */
    double*** d1_u,
    double *** d0_u_next,  /* 3D matrix "Cube" to hold new temperature estimates */
    int N,                /* #nr. interior grid points */
    double d_squared, 
    double inv              ){

    //I AM THE BOTTOM 
    //when highest z value = (N+2)/2 visit my sister device 

    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if(0 < i && 0 < j && 0 < k && k < N+1 && j < N+1 && i < (N+2)/2)
    {    
        if (i == ((N+2)/2)-1) 
        {
            //Peer access
            d0_u_next[i][j][k] = inv * (d0_u[i-1][j][k] + d1_u[0][j][k] + d0_u[i][j-1][k] + d0_u[i][j+1][k] + d0_u[i][j][k-1] + d0_u[i][j][k+1] + d_squared * d0_f[i][j][k]);
        }
        else 
        {
            d0_u_next[i][j][k] = inv * (d0_u[i-1][j][k] + d0_u[i+1][j][k] + d0_u[i][j-1][k] + d0_u[i][j+1][k] + d0_u[i][j][k-1] + d0_u[i][j][k+1] + d_squared * d0_f[i][j][k]);
        }
    }
}

__global__ 
void jacobi_kernel32(
    double*** d1_f,        /* 3D matrix "Cube" of function values, Second derivatives of temperature  */
    double*** d1_u,        /* 3D matrix "Cube" of temperature estimates */
    double*** d0_u,
    double *** d1_u_next,  /* 3D matrix "Cube" to hold new temperature estimates */
    int N,                /* #nr. interior grid points */
    double d_squared, 
    double inv              ){

    //I AM THE TOP
    //when lowest z value = 0 visit my sister device

    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;


    if(0 <= i && 0 < j && 0 < k && k < N+1 && j < N+1 && i < (N+2/2)-1)  //if(0 < i && 0 < j && 0 <= k && i < N+1 && j < N+1 && k < (N+2/2)-1)
    {  
        if (i == 0) 
        {   
            //Peer access
            d1_u_next[i][j][k] = inv * (d0_u[((N+2)/2)-1][j][k] + d1_u[i+1][j][k] + d1_u[i][j-1][k] + d1_u[i][j+1][k] + d1_u[i][j][k-1] + d1_u[i][j][k+1] + d_squared * d1_f[i][j][k]);
        } 
        else 
        {
            d1_u_next[i][j][k] = inv * (d1_u[i-1][j][k] + d1_u[i+1][j][k] + d1_u[i][j-1][k] + d1_u[i][j+1][k] + d1_u[i][j][k-1] + d1_u[i][j][k+1] + d_squared * d1_f[i][j][k]);
        }
    }
}

void jacobi_gpu_wrap3(  double*** d0_f,        /* 3D matrix "Cube" of function values, Second derivatives of temperature  */
                double*** d0_u,        /* 3D matrix "Cube" of temperature estimates */
                double *** d0_u_next,  /* 3D matrix "Cube" to hold new temperature estimates */
                double*** d1_f,        /* 3D matrix "Cube" of function values, Second derivatives of temperature  */
                double*** d1_u,        /* 3D matrix "Cube" of temperature estimates */
                double *** d1_u_next,
                int N,              /* #nr. interior grid points */
                double tolerance,   /* threshold */
                int iter_max,       /* maximum nr. of iterations */
                int * mp){           /* #nr. the iteration needed to get a suciently small diference*/

    double delta= (double)(2.0/((double)(N+1))); 
    double d_squared = delta*delta;
    double inv = 1.0/6.0;
    double *** temp0; // to swipe between u and u_next.
    double *** temp1;
    int m = 0;

    int threads_blck = 8; 

    dim3 dimBlock(threads_blck,threads_blck,threads_blck);// threads per block
    dim3 dimGrid((((N+2)/2)/dimBlock.x)+1,((N+2)/dimBlock.y)+1,((N+2)/dimBlock.z)+1); 

    //printf("Entering while loop\n");
    while (m < iter_max) 
    {
        //DEVICE 0 
        hipSetDevice(0);
        jacobi_kernel31<<<dimGrid,dimBlock>>>(d0_f, d0_u, d1_u, d0_u_next, N, d_squared,inv);     
        
        //DEVICE 1 
        hipSetDevice(1);
        jacobi_kernel32<<<dimGrid,dimBlock>>>(d1_f, d1_u, d0_u, d1_u_next, N, d_squared,inv);    
        checkCudaErrors(hipDeviceSynchronize());  
       
        hipSetDevice(0); 
        checkCudaErrors(hipDeviceSynchronize()); 
 
        temp0 = d0_u;
        d0_u = d0_u_next; 
        d0_u_next = temp0;

        temp1 = d1_u;
        d1_u = d1_u_next; 
        d1_u_next = temp1;
        
        m++;
    }
}



/* *************
 EXERCISE 8 
************* */

__inline__ __device__
double warpReduceSum(double value) 
{ 
    for (int i = 16; i > 0; i /= 2)
    {
        value += __shfl_down_sync(-1, value, i); 
    }
    return value;
}


__inline__ __device__
double blockReduceSum(double value) {
    return warpReduceSum(value);
}


//kernel
__global__ 
void jacobi_kernel4new(
    double*** d_f,        /* 3D matrix "Cube" of function values, Second derivatives of temperature  */
    double*** d_u,        /* 3D matrix "Cube" of temperature estimates */
    double *** d_u_next,  /* 3D matrix "Cube" to hold new temperature estimates */
    int N,                /* #nr. interior grid points */ 
    double inv,
    double d_squared,
    double * norm ){
    double inter_norm;

    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if(0 < i && 0 < j && 0 < k && i < N+1 && j < N+1 && k < N+1)
    {    
        d_u_next[i][j][k] = inv * (d_u[i-1][j][k] + d_u[i+1][j][k] + d_u[i][j-1][k] + d_u[i][j+1][k] + d_u[i][j][k-1] + d_u[i][j][k+1] + d_squared * d_f[i][j][k]);
        inter_norm = (d_u_next[i][j][k] - d_u[i][j][k])*(d_u_next[i][j][k] - d_u[i][j][k]);
        //---- baseline
        //atomicAdd(norm,inter_norm);
        
        //---- v2        
        double value = blockReduceSum(inter_norm);
        if (threadIdx.x == 0){
            atomicAdd(norm, value);
        }

        //---- v3 notes: 
        // for blockReduceSum
        // defined warpSize = 32 
        // lin3axis = blockDim.y*blockDim.z*blockIdx.x + blockIdx.y*blockDim.z + blockIdx.z

    }
    

}

//host warp
void jacobi_gpu_wrap4new(  double*** d_f,   /* 3D matrix "Cube" of function values, Second derivatives of temperature  */
                double*** d_u,              /* 3D matrix "Cube" of temperature estimates */
                double *** d_u_next,        /* 3D matrix "Cube" to hold new temperature estimates */
                int N,                      /* #nr. interior grid points */
                double tolerance,           /* threshold */
                int iter_max,               /* maximum nr. of iterations */
                int * mp){                  /* #nr. the iteration needed to get a suciently small diference*/
    
    double delta= (double)(2.0/((double)(N+1))); // the grid spacing.
    double d_squared = delta*delta;
    double inv = 1.0/6.0; 
    int m = 0; 
    double *** temp; 
    double * d_norm; 
    double * h_norm; 

    hipMalloc((void**)&d_norm,sizeof(double));
    hipHostMalloc((void**)&h_norm,sizeof(double)); 
                    
    int threads_blck = 8; 

    dim3 dimBlock(threads_blck,threads_blck,threads_blck);                          // threads per block
    dim3 dimGrid(((N+2)/dimBlock.x)+1,((N+2)/dimBlock.y)+1,((N+2)/dimBlock.z)+1);   // xx blocks in total

    printf("Calling kernel\n");
    
    *h_norm = 1.0 + tolerance; 

    while(m < iter_max &&  *h_norm > tolerance){
        *h_norm = 0.0;
        hipMemcpy(d_norm, h_norm, sizeof(double),hipMemcpyHostToDevice);
        jacobi_kernel4new<<<dimGrid,dimBlock>>>(d_f, d_u, d_u_next, N,inv,d_squared,d_norm);    
        hipDeviceSynchronize(); 
        
        temp = d_u;
        d_u = d_u_next; 
        d_u_next = temp;

        hipMemcpy(h_norm, d_norm, sizeof(double),hipMemcpyDeviceToHost);
        *h_norm = sqrt(*h_norm);
        //printf("\n h_norm = %e", *h_norm);     
        m++;
    }        

    printf("\n m_break = %d", m-1);
    printf("\n End kernel exercise 8 \n");
    *mp = m-1;
}