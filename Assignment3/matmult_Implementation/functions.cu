extern "C" {

<<<<<<< HEAD
#include "hipblas.h"

void matmult_gpulib(int m,int n,int k,double *A,double *B,double *C){
    
    double *d_A;
    double *d_B;
    double *d_C;

    double alpha = 1.0;
    double beta = 0.0;

    int dimA = k * m*sizeof(double);
    int dimB = k * n*sizeof(double);
    int dimC = n * m*sizeof(double);

    hipMalloc((void **)&d_A, dimA);
    hipMalloc((void **)&d_B, dimB);
    hipMalloc((void **)&d_C, dimC);

    hipblasStatus_t status;

    hipblasHandle_t handle;

    status = hipblasCreate(&handle);
  
    hipMemcpy(d_A, A, dimA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, dimB, hipMemcpyHostToDevice);

    /* Performs operation using cublas */
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, d_B,
                         k, d_A, m, &beta, d_C, m);

    /* Read the result back */
    hipMemcpy(C, d_C, dimC, hipMemcpyDeviceToHost);

    hipFree(d_A); 
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
  }
=======
    #include <stdio.h>     // for in-out purposes
    #include <stdlib.h>    // for memory purposes
    #include "cblas.h"     // for cblas_dgemm Prototype

    /* matmult_lib calls cblas_dgemm from cblas library */
    void matmult_lib(int m,int n,int k,double *A,double *B,double *C) {
        double alpha, beta;
        alpha = 1.0; beta = 0.0;
        cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, m, n, k, alpha, A, k, B, n, beta, C, n);
        
        }


    void matmult_gpulib(int m,int n,int k,double *A,double *B,double *C){
        
    }
    
>>>>>>> 70abe9cc23653bb37dd4facdbb2f5b923f3677ab
    
} // end extern "C"    