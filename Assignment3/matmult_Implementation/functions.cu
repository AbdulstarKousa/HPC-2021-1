#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>    // checkCudaErrors
#include "hipblas.h"      // cublas_dgemm
#include <omp.h>            // parallel, timing , ..etc. 
#include <stdio.h>          // in-out purposes
#include <stdlib.h>         // memory purposes
extern "C" {                // c++ syntax purposes "in matmult_f.nvcc"
#include "cblas.h"          // cblas_dgemm Prototype


void matmult_gpulib(int m,int n,int k,double *A,double *B,double *C){
    
    double *d_A;
    double *d_B;
    double *d_C;

    double alpha = 1.0;
    double beta = 0.0;

    int dimA = k * m*sizeof(double);
    int dimB = k * n*sizeof(double);
    int dimC = n * m*sizeof(double);

    hipMalloc((void **)&d_A, dimA);
    hipMalloc((void **)&d_B, dimB);
    hipMalloc((void **)&d_C, dimC);

    hipblasStatus_t status;

    hipblasHandle_t handle;

    status = hipblasCreate(&handle);

    hipMemcpy(d_A, A, dimA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, dimB, hipMemcpyHostToDevice);

    /* Performs operation using cublas */
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, d_B,
                        n, d_A, k, &beta, d_C, n);

    /* Read the result back */
    hipMemcpy(C, d_C, dimC, hipMemcpyDeviceToHost);

    hipFree(d_A); 
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);

}


/*  matmult_lib:
        calls cblas_dgemm from cblas library, the provided driver(matmult_f.nvcc) will link it to a multithreaded version of CBLAS.
    */
void matmult_lib(int m,int n,int k,double *A,double *B,double *C) {

    double alpha, beta;
    alpha = 1.0; beta = 0.0;
    cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, m, n, k, alpha, A, k, B, n, beta, C, n);
    
    }



//  Two functions for the first sequential implementation of matrix multiplication on the GPU, useing only a single thread:
//         - matmult_gpu1_kernel:  see the comment attached to the function below.
//         - matmult_gpu1:         see the comment attached to the function below. 
    
/*  matmult_gpu1_kernel:  
        helper function, that takes care of the calculations, for the sequential single threaded matmult_gpu1 function.
*/
__global__ void matmult_gpu1_kernel(int m,int n,int k,double *A,double *B,double *C){
    // Matrices Dimensions: 
        // A: m x k 
        // B: k x n
        // C: m x n
    // Here you don't need to think about the columns major as it's one threaded.
    double sum = 0.0;
    for (int i = 0; i != m; i++) {
        for (int j = 0; j != n; j++) {
            sum = 0.0;
            for (int l = 0; l != k; l++) {
                sum += A[i * k +l] * B[l * n  + j];
                // A[i][l] = A[i * k +l] = A[the_current_row(i) * the_number_of_A_columns(k) + the_current_column(l)] 
            }
            C[i * n + j] = sum;
        }
    }
}

/* matmult_gpu1: 
        sequential single threaded function to solve matrxi-matrix multiplication C=AB on the GPU.
*/
void matmult_gpu1(int m,int n,int k,double *A,double *B,double *C){

    // Allocate host memory (here we don't need to allocate host memory as it's already givin as arguments)
    // But as an example bolw is how to Allocate host memory: 
    // double *h_A, *h_B, *h_C;
    // hipHostMalloc((void**)&h_A, m*k*sizeof(double));
    // hipHostMalloc((void**)&h_B, k*n*sizeof(double));
    // hipHostMalloc((void**)&h_C, m*n*sizeof(double));
    
    // Allocate device memory
    double *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, m*k*sizeof(double));
    hipMalloc((void**)&d_B, k*n*sizeof(double));
    hipMalloc((void**)&d_C, m*n*sizeof(double));        

    // Transfer data from host to device memory
    hipMemcpy(d_A, A, m*k*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, k*n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, m*n*sizeof(double), hipMemcpyHostToDevice);

    // Executing kernel 
    matmult_gpu1_kernel<<<1,1>>>(m,n,k,d_A,d_B,d_C); //single threaded (1 block, 1 thread per block)
    checkCudaErrors(hipDeviceSynchronize());

    // Transfer data back to host memory
    hipMemcpy(C, d_C, m*n*sizeof(double), hipMemcpyDeviceToHost);

    // Deallocate device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Deallocate host memory (here we don't need to Deallocate host memory as it was givin as arguments)
    // but as an example bolw is how to Deallocate host memory:
    // hipHostFree(h_A);
    // hipHostFree(h_B);
    // hipHostFree(h_C);        
}



} // end extern "C"    