#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>    // checkCudaErrors command
#include "hipblas.h"      // cublas_dgemm
#include <omp.h>            // parallel, timing , ..etc. 
#include <stdio.h>          // in-out purposes
#include <stdlib.h>         // memory purposes
extern "C" {                // c++ syntax purposes "in matmult_f.nvcc"
#include "cblas.h"          // cblas_dgemm Prototype
#include <assert.h>         // for check 


/*  matmult_lib:
        calls cblas_dgemm from cblas library, the provided driver(matmult_f.nvcc) will link it to a multithreaded version of CBLAS.
*/
void matmult_lib(int m,int n,int k,double *A,double *B,double *C) {
    
    double alpha, beta;
    alpha = 1.0; beta = 0.0;
    cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, m, n, k, alpha, A, k, B, n, beta, C, n);
}


// --------------------------------------------------------------------------
/* Two functions for the first sequential implementation of matrix multiplication on the GPU, using only a single thread:
        - matmult_gpu1_kernel:  see the comment attached to the function below.
        - matmult_gpu1:         see the comment attached to the function below. 
*/


/*  matmult_gpu1_kernel:  
        helper function, that takes care of the calculations, for the sequential single threaded matmult_gpu1 function.
*/
__global__ void matmult_gpu1_kernel(int m,int n,int k,double *A,double *B,double *C){
    // Matrices Dimensions: 
        // A: m x k 
        // B: k x n
        // C: m x n
    // Here you don't need to think about the columns major as it's one threaded.
    double sum = 0.0;
    for (int i = 0; i != m; i++) {
        for (int j = 0; j != n; j++) {
            sum = 0.0;
            for (int l = 0; l != k; l++) {
                sum += A[i * k +l] * B[l * n  + j];
                // A[i][l] = A[i * k +l] = A[the_current_row(i) * the_number_of_A_columns(k) + the_current_column(l)] 
            }
            C[i * n + j] = sum;
        }
    }
}

/* matmult_gpu1: 
        sequential single threaded function to solve matrxi-matrix multiplication, C= A*B, on the GPU.
        Matrices Dimensions: 
            A: m x k 
            B: k x n
            C: m x n
*/
void matmult_gpu1(int m,int n,int k,double *A,double *B,double *C){

    // Allocate host memory (here we don't need to allocate host memory as it's already given as arguments)
    // But as an example below is how to Allocate host memory: 
    // double *h_A, *h_B, *h_C;
    // hipHostMalloc((void**)&h_A, m*k*sizeof(double));
    // hipHostMalloc((void**)&h_B, k*n*sizeof(double));
    // hipHostMalloc((void**)&h_C, m*n*sizeof(double));

    // Allocate device memory
    double *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, m*k*sizeof(double));
    hipMalloc((void**)&d_B, k*n*sizeof(double));
    hipMalloc((void**)&d_C, m*n*sizeof(double));        

    // Transfer data from host to device memory
    hipMemcpy(d_A, A, m*k*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, k*n*sizeof(double), hipMemcpyHostToDevice);

    // Executing kernel 
    matmult_gpu1_kernel<<<1,1>>>(m,n,k,d_A,d_B,d_C); //single threaded (1 block, 1 thread per block)
    checkCudaErrors(hipDeviceSynchronize());

    // Transfer data back to host memory
    hipMemcpy(C, d_C, m*n*sizeof(double), hipMemcpyDeviceToHost);
    
    // Deallocate device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Deallocate host memory (here we don't need to Deallocate host memory as it was given as arguments)
    // but as an example bolw is how to Deallocate host memory:
    // hipHostFree(h_A);
    // hipHostFree(h_B);
    // hipHostFree(h_C);        
}




// --------------------------------------------------------------------------
/* Two functions for gpu2, using one thread per element of C matrix:
        - matmult_gpu2_kernel:  see the comment attached to the function below.
        - matmult_gpu2:         see the comment attached to the function below. 
*/


/*  matmult_gpu2_kernel:  
        helper function, that takes care of the calculations.
*/
__global__ void matmult_gpu2_kernel(int m,int n,int k,double *A,double *B,double *C){

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    double sum =0.0;

    if (i < m && j < n){
        for (int l = 0; l < k; l++) {
            sum += A[i * k +l] * B[l * n + j];
        }
        C[i * n + j] = sum;
    }   
}    

/* matmult_gpu2: 
    Solves C=AB using one thread per element of C matrix
*/
void matmult_gpu2(int m,int n,int k,double *A,double *B,double *C){
    
    double *d_A;
    double *d_B;
    double *d_C;

    int dimA = m * k * sizeof(double);
    int dimB = k * n * sizeof(double);
    int dimC = m * n * sizeof(double);

    hipMalloc((void **)&d_A, dimA);
    hipMalloc((void **)&d_B, dimB);
    hipMalloc((void **)&d_C, dimC);

    hipMemcpy(d_A, A, dimA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, dimB, hipMemcpyHostToDevice);

    int numThreads = 32;

    dim3 threadsPerBlock = dim3(numThreads,numThreads);
    dim3 blocks = dim3(n/numThreads+1, m/numThreads+1);

    matmult_gpu2_kernel<<<blocks,threadsPerBlock>>>(m, n, k, d_A, d_B, d_C);
    hipDeviceSynchronize();

    hipMemcpy(C, d_C, dimC, hipMemcpyDeviceToHost);

    hipFree(d_A); 
    hipFree(d_B);
    hipFree(d_C);   
}

// --------------------------------------------------------------------------
/* Two functions for gpu3, where each thread computes exactly two elements of C matrix:
        - matmult_gpu3_kernel:  see the comment attached to the function below.
        - matmult_gpu3:         see the comment attached to the function below. 
*/

/*  matmult_gpu3_kernel:  
        helper function, that takes care of the calculations.
*/
__global__ void matmult_gpu3_kernel(int m,int n,int k,double *A,double *B,double *C){

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = 2*(blockIdx.y * blockDim.y + threadIdx.y);

    double sum1 =0.0;
    double sum2 =0.0;

    if (i < m && j < n){
        for (int l = 0; l < k; l++) {
            sum1 += A[i * k +l] * B[l * n + j];
            if (i+1 < m){
                sum2 += A[(i+1) * k +l] * B[l * n + j];
            }             
        }
        C[i * n + j] = sum1;
        if (i+1 < m) {
            C[(i+1) * n + j] = sum2;
        }
    }   
}    


/* matmult_gpu3: 
    Solves C=AB where each thread computes exactly two elements of C matrix.
*/
void matmult_gpu3(int m,int n,int k,double *A,double *B,double *C){

    double *d_A;
    double *d_B;
    double *d_C;

    int dimA = m * k * sizeof(double);
    int dimB = k * n * sizeof(double);
    int dimC = m * n * sizeof(double);

    hipMalloc((void **)&d_A, dimA);
    hipMalloc((void **)&d_B, dimB);
    hipMalloc((void **)&d_C, dimC);

    hipMemcpy(d_A, A, dimA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, dimB, hipMemcpyHostToDevice);

    int numThreads = 16;

    dim3 threadsPerBlock = dim3(numThreads,numThreads);
    dim3 blocks = dim3(n/numThreads+1, m/(numThreads*2)+1);

    matmult_gpu3_kernel<<<blocks,threadsPerBlock>>>(m, n, k, d_A, d_B, d_C);

    hipDeviceSynchronize();

    hipMemcpy(C, d_C, dimC, hipMemcpyDeviceToHost);

    hipFree(d_A); 
    hipFree(d_B);
    hipFree(d_C);

}

// __global__ void matmult_gpu3_col_kernel(int m,int n,int k,double *A,double *B,double *C){

//     int j = 2*(blockIdx.x * blockDim.x + threadIdx.x);
//     int i = blockIdx.y * blockDim.y + threadIdx.y;

//     double sum1 =0.0;
//     double sum2 =0.0;

//     if (i < m && j < n){
//         for (int l = 0; l < k; l++) {
//             sum1 += A[i * k +l] * B[l * n + j];
//             if (j+1 < n) 
//             sum2 += A[i * k +l] * B[l * n + (j+1)];
//         }
//         C[i * n + j] = sum1;
//         if (j+1 < n) 
//         C[i * n + (j+1)] = sum2;
//     }   
// }    

// void matmult_gpu3_col(int m,int n,int k,double *A,double *B,double *C){
    
//     double *d_A;
//     double *d_B;
//     double *d_C;

//     int dimA = m * k * sizeof(double);
//     int dimB = k * n * sizeof(double);
//     int dimC = m * n * sizeof(double);

//     hipMalloc((void **)&d_A, dimA);
//     hipMalloc((void **)&d_B, dimB);
//     hipMalloc((void **)&d_C, dimC);

//     hipMemcpy(d_A, A, dimA, hipMemcpyHostToDevice);
//     hipMemcpy(d_B, B, dimB, hipMemcpyHostToDevice);

//     int numThreads = 16;

//     dim3 threadsPerBlock = dim3(numThreads,numThreads);

//     dim3 blocks = dim3(n/(numThreads*2)+1, m/numThreads+1);

//     matmult_gpu3_col_kernel<<<blocks,threadsPerBlock>>>(m, n, k, d_A, d_B, d_C);

//     hipDeviceSynchronize();

//     hipMemcpy(C, d_C, dimC, hipMemcpyDeviceToHost);

//     hipFree(d_A); 
//     hipFree(d_B);
//     hipFree(d_C);

// }

// __global__ void matmult_gpu4_4_kernel(int m,int n,int k,double *A,double *B,double *C){


//     int j = blockIdx.x * blockDim.x + threadIdx.x;
//     int i = 4*(blockIdx.y * blockDim.y + threadIdx.y);

//     double sum0 =0.0;
//     double sum1 =0.0;
//     double sum2 =0.0;
//     double sum3 =0.0;

//     if (i < m && j < n){
//         for (int l = 0; l < k; l++) {
//             sum0 += A[i * k +l] * B[l * n + j];
//             if (i+1 < m){
//                 sum1 += A[(i+1) * k +l] * B[l * n + j];
//             } 
//             if (i+2 < m){
//                 sum2 += A[(i+2) * k +l] * B[l * n + j];
//             }  
//             if (i+3 < m){
//                 sum3 += A[(i+3) * k +l] * B[l * n + j];
//             }              
//         }
//         C[i * n + j] = sum0;
//         if (i+1 < m) {
//             C[(i+1) * n + j] = sum1;
//         }
//         if (i+2 < m) {
//             C[(i+2) * n + j] = sum2;
//         }
//         if (i+3 < m) {
//             C[(i+3) * n + j] = sum3;
//         }
//     }    
// }    

// void matmult_gpu4_4(int m,int n,int k,double *A,double *B,double *C){
    
//     double *d_A;
//     double *d_B;
//     double *d_C;

//     int dimA = m * k * sizeof(double);
//     int dimB = k * n * sizeof(double);
//     int dimC = m * n * sizeof(double);

//     hipMalloc((void **)&d_A, dimA);
//     hipMalloc((void **)&d_B, dimB);
//     hipMalloc((void **)&d_C, dimC);

//     hipMemcpy(d_A, A, dimA, hipMemcpyHostToDevice);
//     hipMemcpy(d_B, B, dimB, hipMemcpyHostToDevice);

//     int numThreads = 32;

//     dim3 threadsPerBlock = dim3(numThreads,numThreads);

//     dim3 blocks = dim3(n/numThreads+1, m/(numThreads*4)+1);

//     matmult_gpu4_4_kernel<<<blocks,threadsPerBlock>>>(m, n, k, d_A, d_B, d_C);

//     hipDeviceSynchronize();

//     hipMemcpy(C, d_C, dimC, hipMemcpyDeviceToHost);

//     hipFree(d_A); 
//     hipFree(d_B);
//     hipFree(d_C);
// }



// --------------------------------------------------------------------------
/* Two functions for gpu4, where each thread computes > 2 elements of C:
        - matmult_gpu4_kernel:  see the comment attached to the function below.
        - matmult_gpu4:         see the comment attached to the function below. 
*/


/*  matmult_gpu4_kernel:  
        helper function, that takes care of the calculations.
*/
__global__ void matmult_gpu4_kernel(int m,int n,int k,double *A,double *B,double *C){


    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = 8*(blockIdx.y * blockDim.y + threadIdx.y);

    double sum0 =0.0;
    double sum1 =0.0;
    double sum2 =0.0;
    double sum3 =0.0;
    double sum4 =0.0;
    double sum5 =0.0;
    double sum6 =0.0;
    double sum7 =0.0;

    if (i < m && j < n){
        for (int l = 0; l < k; l++) {
            sum0 += A[i * k +l] * B[l * n + j];
            if (i+1 < m){
                sum1 += A[(i+1) * k +l] * B[l * n + j];
            } 
            if (i+2 < m){
                sum2 += A[(i+2) * k +l] * B[l * n + j];
            }  
            if (i+3 < m){
                sum3 += A[(i+3) * k +l] * B[l * n + j];
            } 
            if (i+4 < m){
                sum4 += A[(i+4) * k +l] * B[l * n + j];
            } 
            if (i+5 < m){
                sum5 += A[(i+5) * k +l] * B[l * n + j];
            }  
            if (i+6 < m){
                sum6 += A[(i+6) * k +l] * B[l * n + j];
            }     
            if (i+7 < m){
                sum7 += A[(i+7) * k +l] * B[l * n + j];
            }          
        }
        C[i * n + j] = sum0;
        if (i+1 < m) {
            C[(i+1) * n + j] = sum1;
        }
        if (i+2 < m) {
            C[(i+2) * n + j] = sum2;
        }
        if (i+3 < m) {
            C[(i+3) * n + j] = sum3;
        }
        if (i+4 < m) {
            C[(i+4) * n + j] = sum4;
        }
        if (i+5 < m) {
            C[(i+5) * n + j] = sum5;
        }
        if (i+6 < m) {
            C[(i+6) * n + j] = sum6;
        }
        if (i+7 < m) {
            C[(i+7) * n + j] = sum7;
        }
    }    
}    


/* matmult_gpu4: 
    Solves C=AB where each thread computes > 2 elements of C.
*/
void matmult_gpu4(int m,int n,int k,double *A,double *B,double *C){

    double *d_A;
    double *d_B;
    double *d_C;

    int dimA = m * k * sizeof(double);
    int dimB = k * n * sizeof(double);
    int dimC = m * n * sizeof(double);

    hipMalloc((void **)&d_A, dimA);
    hipMalloc((void **)&d_B, dimB);
    hipMalloc((void **)&d_C, dimC);

    hipMemcpy(d_A, A, dimA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, dimB, hipMemcpyHostToDevice);

    int numThreads = 32;

    dim3 threadsPerBlock = dim3(numThreads,numThreads);

    dim3 blocks = dim3(n/numThreads+1, m/(numThreads*8)+1);

    matmult_gpu4_kernel<<<blocks,threadsPerBlock>>>(m, n, k, d_A, d_B, d_C);

    hipDeviceSynchronize();

    hipMemcpy(C, d_C, dimC, hipMemcpyDeviceToHost);

    hipFree(d_A); 
    hipFree(d_B);
    hipFree(d_C);
}



// --------------------------------------------------------------------------
/* Two functions for gpu5 that uses shared memory for reading the A and B matrices in order to improve the performance:
        - matmult_gpu5_kernel:  see the comment attached to the function below.
        - matmult_gpu5:         see the comment attached to the function below. 
    Please make sure when you run matmult_gpu5 that m, n and k are integer multiples of the thread block size = 16.
*/

// Thread block size
#define BLOCK_SIZE 16 
#define INPUT_ERR fprintf(stderr,"%s:\nOne or more of the defiend values for m , n , k are not integer multiples of the thread block size = %d.\n",__func__,BLOCK_SIZE)
/*  matmult_gpu5_kernel:  
        helper function, that takes care of the calculations.
*/
__global__ void matmult_gpu5_kernel(int m,int n,int k,double *A,double *B,double *C){
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    double * Csub;
    Csub = &C[n* BLOCK_SIZE * blockRow + BLOCK_SIZE * blockCol];

    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    double Cvalue = 0.0;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
    for (int i = 0; i < (k / BLOCK_SIZE); ++i) {

        // Get sub-matrix Asub of A
        double * Asub; 
        Asub = &A[k * BLOCK_SIZE * blockRow + BLOCK_SIZE * i]; 

        // Get sub-matrix Bsub of B
        double * Bsub; 
        Bsub = &B[n* BLOCK_SIZE * i + BLOCK_SIZE * blockCol];  

        // Shared memory used to store Asub and Bsub respectively
        __shared__ double As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = Asub[row * k + col]; // As[row][col] = A[row * k + col + k * BLOCK_SIZE * blockRow + BLOCK_SIZE * i];
        Bs[row][col] = Bsub[row * n + col]; // Bs[row][col] = B[row * n + col + n* BLOCK_SIZE * i + BLOCK_SIZE * blockCol];

        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();

        // Multiply Asub and Bsub together
        for (int e = 0; e < BLOCK_SIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write Csub to device memory
    // Each thread writes one element
    Csub[row * n + col] = Cvalue;  // C[row * n + col + n* BLOCK_SIZE * blockRow + BLOCK_SIZE * blockCol] = Cvalue;

}


/* matmult_gpu5: 
    Solves C=AB useing shared memory for reading the A and B matrices in order to improve the performance
*/
void matmult_gpu5(int m,int n,int k,double *A,double *B,double *C){

    

    //making sure that m, n and k are integer multiples of the thread block size.
    if(m%BLOCK_SIZE!=0 || n%BLOCK_SIZE!=0 || k%BLOCK_SIZE!=0){
        INPUT_ERR;
        assert(m%BLOCK_SIZE!=0 == 0);
        assert(n%BLOCK_SIZE!=0 == 0);
        assert(k%BLOCK_SIZE!=0 == 0);
    }

    // Allocate device memory
    double *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, m*k*sizeof(double));
    hipMalloc((void**)&d_B, k*n*sizeof(double));
    hipMalloc((void**)&d_C, m*n*sizeof(double));

    // Transfer data from host to device memory
    hipMemcpy(d_A, A, m*k*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, k*n*sizeof(double), hipMemcpyHostToDevice);


    // Executing kernel 
    // For simplicity,  it is assumed that m, n and k are integer multiples of the thread block size = 16. See Assignment task. 
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(n / dimBlock.x, m / dimBlock.y);

    matmult_gpu5_kernel<<<dimGrid, dimBlock>>>(m,n,k,d_A,d_B,d_C); 
    checkCudaErrors(hipDeviceSynchronize());

    // Transfer data back to host memory
    hipMemcpy(C, d_C, m*n*sizeof(double), hipMemcpyDeviceToHost);

    // Deallocate device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

// Thread block size
#define BLOCK_SIZE1 32 
#define INPUT_ERR fprintf(stderr,"%s:\nOne or more of the defiend values for m , n , k are not integer multiples of the thread block size = %d.\n",__func__,BLOCK_SIZE)
/*  matmult_gpu5_kernel:  
        helper function, that takes care of the calculations.
*/
__global__ void matmult_gpu6_kernel(int m,int n,int k,double *A,double *B,double *C){
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    double * Csub;
    Csub = &C[n* BLOCK_SIZE1 * blockRow + BLOCK_SIZE1 * blockCol];

    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    double Cvalue = 0.0;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
    for (int i = 0; i < (k / BLOCK_SIZE1); ++i) {

        // Get sub-matrix Asub of A
        double * Asub; 
        Asub = &A[k * BLOCK_SIZE1 * blockRow + BLOCK_SIZE1 * i]; 

        // Get sub-matrix Bsub of B
        double * Bsub; 
        Bsub = &B[n* BLOCK_SIZE1 * i + BLOCK_SIZE1 * blockCol];  

        // Shared memory used to store Asub and Bsub respectively
        __shared__ double As[BLOCK_SIZE1][BLOCK_SIZE1];
        __shared__ double Bs[BLOCK_SIZE1][BLOCK_SIZE1];

        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = Asub[row * k + col]; // As[row][col] = A[row * k + col + k * BLOCK_SIZE * blockRow + BLOCK_SIZE * i];
        Bs[row][col] = Bsub[row * n + col]; // Bs[row][col] = B[row * n + col + n* BLOCK_SIZE * i + BLOCK_SIZE * blockCol];

        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();

        // Multiply Asub and Bsub together
        for (int e = 0; e < BLOCK_SIZE1; ++e)
            Cvalue += As[row][e] * Bs[e][col];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write Csub to device memory
    // Each thread writes one element
    Csub[row * n + col] = Cvalue;  // C[row * n + col + n* BLOCK_SIZE * blockRow + BLOCK_SIZE * blockCol] = Cvalue;

}


/* matmult_gpu5: 
    Solves C=AB useing shared memory for reading the A and B matrices in order to improve the performance
*/
void matmult_gpu6(int m,int n,int k,double *A,double *B,double *C){

    

    //making sure that m, n and k are integer multiples of the thread block size.
    if(m%BLOCK_SIZE1!=0 || n%BLOCK_SIZE1!=0 || k%BLOCK_SIZE1!=0){
        INPUT_ERR;
        assert(m%BLOCK_SIZE1!=0 == 0);
        assert(n%BLOCK_SIZE1!=0 == 0);
        assert(k%BLOCK_SIZE1!=0 == 0);
    }

    // Allocate device memory
    double *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, m*k*sizeof(double));
    hipMalloc((void**)&d_B, k*n*sizeof(double));
    hipMalloc((void**)&d_C, m*n*sizeof(double));

    // Transfer data from host to device memory
    hipMemcpy(d_A, A, m*k*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, k*n*sizeof(double), hipMemcpyHostToDevice);


    // Executing kernel 
    // For simplicity,  it is assumed that m, n and k are integer multiples of the thread block size = 16. See Assignment task. 
    dim3 dimBlock(BLOCK_SIZE1, BLOCK_SIZE1);
    dim3 dimGrid(n / dimBlock.x, m / dimBlock.y);

    matmult_gpu6_kernel<<<dimGrid, dimBlock>>>(m,n,k,d_A,d_B,d_C); 
    checkCudaErrors(hipDeviceSynchronize());

    // Transfer data back to host memory
    hipMemcpy(C, d_C, m*n*sizeof(double), hipMemcpyDeviceToHost);

    // Deallocate device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

/*  matmult_gpulib:
    calls DGEMM function for GPUs provided by Nvidia in the CUBLAS library        
*/
void matmult_gpulib(int m,int n,int k,double *A,double *B,double *C){

    double *d_A;
    double *d_B;
    double *d_C;

    double alpha = 1.0;
    double beta = 0.0;

    int dimA = k * m*sizeof(double);
    int dimB = k * n*sizeof(double);
    int dimC = n * m*sizeof(double);

    hipMalloc((void **)&d_A, dimA);
    hipMalloc((void **)&d_B, dimB);
    hipMalloc((void **)&d_C, dimC);

    hipblasStatus_t status;

    hipblasHandle_t handle;

    status = hipblasCreate(&handle);

    hipMemcpy(d_A, A, dimA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, dimB, hipMemcpyHostToDevice);

    /* Performs operation using cublas */
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, d_B,
        n, d_A, k, &beta, d_C, n);

    /* Read the result back */
    hipMemcpy(C, d_C, dimC, hipMemcpyDeviceToHost);

    hipFree(d_A); 
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
}


} // end extern "C"    