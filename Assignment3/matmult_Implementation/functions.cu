
#include "hipblas.h"

void matmult_gpulib(int m,int n,int k,double *A,double *B,double *C){
    
    double *d_A;
    double *d_B;
    double *d_C;

    double alpha = 1.0;
    double beta = 0.0;

    int dimA = k * m*sizeof(double);
    int dimB = k * n*sizeof(double);
    int dimC = n * m*sizeof(double);

    hipMalloc((void **)&d_A, dimA);
    hipMalloc((void **)&d_B, dimB);
    hipMalloc((void **)&d_C, dimC);

    hipblasStatus_t status;

    hipblasHandle_t handle;

    status = hipblasCreate(&handle);
  
    hipMemcpy(d_A, A, dimA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, dimB, hipMemcpyHostToDevice);

    /* Performs operation using cublas */
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, d_B,
                         k, d_A, m, &beta, d_C, m);

    /* Read the result back */
    hipMemcpy(C, d_C, dimC, hipMemcpyDeviceToHost);

    hipFree(d_A); 
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
  }
    
}